#include "hip/hip_runtime.h"
#include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>
#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/blas.h>

#include <iostream>
#include <string>
#include <map>
#include <cmath>
#include <stdio.h>

#include "mkl.h"
#include "../timer.h"

template <typename IndexType, typename ValueType>
void benchmark_mkl_spmv(cusp::csr_matrix<IndexType, ValueType, cusp::host_memory>& A)
{
    cusp::array1d<ValueType,cusp::host_memory> x(A.num_cols, 0);
    cusp::array1d<ValueType,cusp::host_memory> y(A.num_rows, 0);

    // warm up
    spmv_mkl_double(A.num_rows,
                    &A.values[0],
                    &A.row_offsets[0],
                    &A.column_indices[0],
                    &x[0],
                    &y[0]);

    // benchmark SpMV
    timer t;

    const size_t num_iterations = 500;

    for(size_t i = 0; i < num_iterations; i++)
        spmv_mkl_double(A.num_rows,
                        &A.values[0],
                        &A.row_offsets[0],
                        &A.column_indices[0],
                        &x[0],
                        &y[0]);


    float time = t.seconds_elapsed() / num_iterations;
    float GFLOPs = (time == 0) ? 0 : (2 * A.num_entries / time) / 1e9;

    printf("MKL SpMV %8.4f ms ( %5.2f GFLOP/s )\n", 1e3 * time, GFLOPs); 
}
template <typename IndexType, typename ValueType>
void benchmark_cusp_spmv(cusp::csr_matrix<IndexType, ValueType, cusp::host_memory>& A_host)
{
    cusp::hyb_matrix<IndexType,ValueType,cusp::device_memory> A(A_host);

    cusp::array1d<ValueType,cusp::device_memory> x(A.num_cols, 0);
    cusp::array1d<ValueType,cusp::device_memory> y(A.num_rows, 0);
    
    // warm up
    A.multiply(x, y);

    // benchmark SpMV
    timer t;

    const size_t num_iterations = 500;

    for(size_t i = 0; i < num_iterations; i++)
        A.multiply(x, y);

    float time = t.seconds_elapsed() / num_iterations;
    float GFLOPs = (time == 0) ? 0 : (2 * A.num_entries / time) / 1e9;

    printf("CUSP SpMV %8.4f ms ( %5.2f GFLOP/s )\n", 1e3 * time, GFLOPs); 
}

template <typename IndexType, typename ValueType>
void benchmark_mkl_cg(cusp::csr_matrix<IndexType, ValueType, cusp::host_memory>& A,
                      cusp::array1d<ValueType, cusp::host_memory>& b,
                      ValueType target_residual)
{
    cusp::array1d<ValueType, cusp::host_memory> x(A.num_rows, 0);

    timer t;

    cg_mkl_double(A.num_rows,
                  &A.values[0],
                  &A.row_offsets[0],
                  &A.column_indices[0],
                  &x[0],
                  &b[0],
                  A.num_rows,
                  ValueType(0),
                  target_residual);

    float time = t.seconds_elapsed();

    printf("MKL CG finished in %8.4f secs\n", time);
}


template <typename IndexType, typename ValueType>
void benchmark_cusp_cg(cusp::csr_matrix<IndexType, ValueType, cusp::host_memory>& A_host,
                       cusp::array1d<ValueType, cusp::host_memory>& b_host,
                       ValueType target_residual)
{
    cusp::hyb_matrix<IndexType,ValueType,cusp::device_memory> A(A_host);

    cusp::array1d<ValueType, cusp::device_memory> x(A.num_rows, 0);
    cusp::array1d<ValueType, cusp::device_memory> b(b_host);

    // set stopping criteria:
    cusp::default_stopping_criteria stopping_criteria(target_residual/cusp::blas::nrm2(b), A.num_rows);

    // set preconditioner (identity)
    cusp::identity_operator<ValueType, cusp::device_memory> M(A.num_rows, A.num_rows);

    // set verbosity level
    int verbose = 1;
    
    timer t;

    // obtain a linear operator from matrix A and call CG
    cusp::krylov::cg(A, x, b, stopping_criteria, M, verbose);

    float time = t.seconds_elapsed();

    printf("CUSP CG finished in %8.4f secs\n", time);
}

int main(void)
{
    hipSetDevice(1);

    typedef int    IndexType;
    typedef double ValueType;

    cusp::csr_matrix<IndexType, ValueType, cusp::host_memory> A;
    cusp::array1d<ValueType, cusp::host_memory> x;
    cusp::array1d<ValueType, cusp::host_memory> b;

    cusp::io::read_matrix_market_file(A, "A.mtx");
    { cusp::array2d<ValueType, cusp::host_memory> temp; cusp::read_matrix_market_file(temp, "x.mtx"); temp.values.swap(x); }
    { cusp::array2d<ValueType, cusp::host_memory> temp; cusp::read_matrix_market_file(temp, "b.mtx"); temp.values.swap(b); }
    
    std::cout << "loaded matrix with shape (" << A.num_rows << "," << A.num_cols << ") and " << A.num_entries << " entries" << "\n\n";

    std::cout << "---------- benchmarking SpMV ----------\n";
    benchmark_mkl_spmv(A);
    benchmark_cusp_spmv(A);

    if (A.num_rows == A.num_cols)
    {
        std::cout << "\n----------- benchmarking CG -----------\n";
        // compute residual
        cusp::array1d<ValueType, cusp::host_memory> r(A.num_rows,0);
        A.multiply(x,r);
        cusp::blas::axpy(b, r, ValueType(-1.0));

        ValueType residual_norm = cusp::blas::nrm2(r);
        
        std::cout << " provided solution has residual norm " << residual_norm << std::endl;

        benchmark_mkl_cg(A, b, residual_norm);
        benchmark_cusp_cg(A, b, residual_norm);
    }

    return 0;
}

